#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "get_walltime.c"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"%s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void array_print(float *arr, int length) 
{
  for (int i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}



__global__ void bitonicSort(float *GPU_array, int j, int i)
{
  unsigned int index, swap_index; 
  index = threadIdx.x + blockDim.x * blockIdx.x;
  swap_index = index^j;

  if ((swap_index)>index) {
    if ((index&i)==0 && GPU_array[index]>GPU_array[swap_index]) {
      float temp = GPU_array[index];
      GPU_array[index] = GPU_array[swap_index];
      GPU_array[swap_index] = temp;
    }
    if ((index&i)!=0 && GPU_array[index]<GPU_array[swap_index]) {
      float temp = GPU_array[index];
      GPU_array[index] = GPU_array[swap_index];
      GPU_array[swap_index] = temp;
    }
  }
}

int main(void)
{
    int N;
    double start=0, stop=0;
    
    scanf("%d", &N);

    int num_threads = 32;
    int num_blocks = (N+num_threads-1)+num_threads;
    float *array = new float[N];
    float *GPU_array;
    size_t GPU_N = N * sizeof(float);

    srand(time(NULL));
    float a = 10.0;

    for (int i = 0; i < N; i++){
      array[i] = float((rand())/float((RAND_MAX)) * a);
    }

    //array_print(array, N);

    get_walltime(&start); 
    
    hipMalloc((void**) &GPU_array, GPU_N);
    gpuErrchk( hipMemcpy(GPU_array, array, GPU_N, hipMemcpyHostToDevice) );
    hipMemcpy(GPU_array, array, GPU_N, hipMemcpyHostToDevice);
    
    

    dim3 blocks(num_blocks,1);    
    dim3 threads(num_threads,1);  

    
    for (int i = 2; i <= N; i *= 2) {
      for (int j=i>>1; j>0; j=j>>1) {
        bitonicSort<<<blocks, threads>>>(GPU_array, j, i);
        hipDeviceSynchronize();
      }
    }
    
    hipMemcpy(array, GPU_array, GPU_N, hipMemcpyDeviceToHost);
    
    hipFree(GPU_array);

    //array_print(array, N);
    get_walltime(&stop);

    
    printf("Time taken for array size %d with %d threads: %.3fs\n", N, num_threads, stop-start);
 
}